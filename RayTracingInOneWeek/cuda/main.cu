#include <iostream>
#include <hip/hip_runtime.h>

__global__ void render(float *img, int max_x, int max_y) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    if((i >= max_x) || (j >= max_y)) return;

    float r = float(i) / (max_x - 1);
    float g = float(j) / (max_y - 1);
    float b = 0.25;

    int pixel_index = j * max_x * 3 + i * 3;
    img[pixel_index + 0] = r;
    img[pixel_index + 1] = g;
    img[pixel_index + 2] = b;
}

int main() {
    const int img_wid = 256;
    const int img_height = 256;
    const int img_size = img_wid * img_height * 3;
    float *img_buffer;

    hipMallocManaged(&img_buffer, img_size * sizeof(float));

    dim3 blocks(img_wid / 16, img_height / 16);
    dim3 threads(16, 16);

    render<<<blocks, threads>>>(img_buffer, img_wid, img_height);

    hipDeviceSynchronize();

    std::cout << "P3\n" << img_wid << ' ' << img_height << "\n255\n";
    for(int j = 0; j < img_height; ++j) {
        for(int i = 0; i < img_wid; ++i) {
            int pixel_index = j * img_wid * 3 + i * 3;
            float r = img_buffer[pixel_index + 0];
            float g = img_buffer[pixel_index + 1];
            float b = img_buffer[pixel_index + 2];

            int ir = static_cast<int>(255.999 * r);
            int ig = static_cast<int>(255.999 * g);
            int ib = static_cast<int>(255.999 * b);

            std::cout << ir << ' ' << ig << ' ' << ib << '\n';
        }
    }

    hipFree(img_buffer);
    return 0;
}
