#include <iostream>
#include <hip/hip_runtime.h>
#include <time.h>

__global__ void render(float *img, int max_x, int max_y) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    if((i >= max_x) || (j >= max_y)) return;

    float r = float(i) / (max_x - 1);
    float g = float(j) / (max_y - 1);
    float b = 0.25;

    int pixel_index = j * max_x * 3 + i * 3;
    img[pixel_index + 0] = r;
    img[pixel_index + 1] = g;
    img[pixel_index + 2] = b;
}

int main() {
    const int img_wid = 256;
    const int img_height = 256;
    const int img_size = img_wid * img_height * 3;
    float *img_buffer;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipMallocManaged(&img_buffer, img_size * sizeof(float));

    // Start record
    hipEventRecord(start);
    dim3 blocks(img_wid / 16, img_height / 16);
    dim3 threads(16, 16);
    
    render<<<blocks, threads>>>(img_buffer, img_wid, img_height);

    hipDeviceSynchronize();
    
    // Stop record
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    // Calculate elapsed time
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    std::cerr << "Image generation took " << milliseconds << " milliseconds.\n";


    std::cout << "P3\n" << img_wid << ' ' << img_height << "\n255\n";
    for(int j = 0; j < img_height; ++j) {
        for(int i = 0; i < img_wid; ++i) {
            int pixel_index = j * img_wid * 3 + i * 3;
            float r = img_buffer[pixel_index + 0];
            float g = img_buffer[pixel_index + 1];
            float b = img_buffer[pixel_index + 2];

            int ir = static_cast<int>(255.999 * r);
            int ig = static_cast<int>(255.999 * g);
            int ib = static_cast<int>(255.999 * b);

            std::cout << ir << ' ' << ig << ' ' << ib << '\n';
        }
    }

    hipFree(img_buffer);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    return 0;
}
